#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {






template <typename Dtype>
__global__ void ReLUForward(const int n, const Dtype* in, Dtype* out,
    Dtype negative_slope,const Dtype* gates,Dtype* activation_data,bool activation_on) {
  CUDA_KERNEL_LOOP(index, n) {
    if(activation_on){
    out[index] = activation_data[index]*gates[index]*(in[index] > 0 ? in[index] : in[index] * negative_slope);
    }else{
   
    out[index] = gates[index]*(in[index] > 0 ? in[index] : in[index] * negative_slope);
    }
  }
}

template <typename Dtype>
void ReLULayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const Dtype* gates_data = this->blobs_[0]->gpu_data();
  Dtype* buffer_data = this->blobs_[2]->mutable_gpu_data();
  const int count = bottom[0]->count();
  Dtype negative_slope = this->layer_param_.relu_param().negative_slope();
  Dtype* activation_data =this->blobs_[1]->mutable_gpu_data();
  // NOLINT_NEXT_LINE(whitespace/operators)
  ReLUForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data, negative_slope,gates_data,activation_data,activation_on);
  CUDA_POST_KERNEL_CHECK;
    caffe_copy(count,top_data,buffer_data);
  // << " count: " << count << " bottom_data: "
  //     << (unsigned long)bottom_data
  //     << " top_data: " << (unsigned long)top_data
  //     << " blocks: " << CAFFE_GET_BLOCKS(count)
  //     << " threads: " << CAFFE_CUDA_NUM_THREADS;
}

template <typename Dtype>
__global__ void ReLUBackward(const int n, const Dtype* in_diff,
    const Dtype* in_data, Dtype* out_diff, Dtype negative_slope,const Dtype* gates, Dtype* activation_data ,bool activation_on) {
   if(activation_on){
       CUDA_KERNEL_LOOP(index, n) {
           activation_data[index]=in_diff[index]>0?Dtype(1.0):Dtype(0.0);
           out_diff[index] = activation_data[index]*gates[index]*(in_diff[index] * ((in_data[index]>0)
           + (in_data[index] <= 0) * negative_slope));}
        

   }else{
        CUDA_KERNEL_LOOP(index, n) {
           out_diff[index] = gates[index]*(in_diff[index] * ((in_data[index] > 0)
           + (in_data[index] <= 0) * negative_slope));}
        }  
}


template <typename Dtype>
__global__ void Threshold1(const int n, const Dtype threshold,
    const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] > threshold ? in[index] : 0;
  }
}

template <typename Dtype>
__global__ void Threshold2(const int n, const Dtype threshold,
    const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] > threshold ? 1 : 0;
  }
}

template <typename Dtype>
__global__ void Threshold3(const int n, const Dtype threshold,
    const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] >= threshold ? 1 : 0;
  }
}

template <typename Dtype>
__global__ void drop_edge_pixels(const int count, const int n,const int c, const int h,const int w,
    const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, count) {
   
   int num=index/(c*h*w);
   int ch=(index-num*c*h*w)/(h*w);
   int hei=(index-num*c*h*w-ch*h*w)/h;
   int wid=index-num*c*h*w-ch*h*w-hei*w;
   if (hei==0||hei==h-1||wid==0||wid==w-1){
   
    out[index] =0;
   }else{
    out[index]=in[index];
   }

  }
}





template <typename Dtype>
void ReLULayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* top_diff = top[0]->mutable_gpu_diff();
    const Dtype* gates_data = this->blobs_[0]->gpu_data();
    Dtype* mutable_gates_data = this->blobs_[0]->mutable_gpu_data();
    Dtype* gates_diff = this->blobs_[0]->mutable_gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    Dtype negative_slope = this->layer_param_.relu_param().negative_slope();
    Dtype diff_nonezeros=0;
    Dtype diff_mean=0;
    Dtype threshold=0;   
   // Dtype* activation_data = activation.mutable_gpu_data();
   Dtype* activation_data =this->blobs_[1]->mutable_gpu_data();
    if(this->bp_mode==0){
      ReLUBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
            count, top_diff, bottom_data, bottom_diff, negative_slope,gates_data,activation_data,activation_on);
      CUDA_POST_KERNEL_CHECK;




    }
    if(this->bp_mode==1){

       if (this->blobs_[0]->height()>1&&this->blobs_[0]->width()>1){
           // drop edge pixels
         
           drop_edge_pixels<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count,this->blobs_[0]->num(),this->blobs_[0]->channels(),this->blobs_[0]->height(),this->blobs_[0]->width(),top_diff, top_diff);
           CUDA_POST_KERNEL_CHECK; 
       }




    //nonezeros' mean
         Threshold2<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, Dtype(0), top_diff, gates_diff);
    CUDA_POST_KERNEL_CHECK;

    caffe_gpu_asum(count,gates_diff,&diff_nonezeros);

    Threshold1<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, Dtype(0), top_diff, gates_diff);
    CUDA_POST_KERNEL_CHECK;

    caffe_gpu_asum(count,gates_diff,&diff_mean);
    
    diff_mean=diff_mean/diff_nonezeros;
    //threshold
    threshold=this->threshold_ratio*diff_mean;

   
    Threshold3<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, threshold, gates_diff, mutable_gates_data);

    ReLUBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
            count, top_diff, bottom_data, bottom_diff, negative_slope,gates_data,activation_data,activation_on);
      CUDA_POST_KERNEL_CHECK;
    
   }
    if(this->bp_mode==2){
    // NOLINT_NEXT_LINE(whitespace/operators)
    ReLUBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, bottom_data, bottom_diff, negative_slope,gates_data,activation_data,activation_on);
    CUDA_POST_KERNEL_CHECK;

 
       if (this->blobs_[0]->height()>1&&this->blobs_[0]->width()>1){
           // drop edge pixels
         
           drop_edge_pixels<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count,this->blobs_[0]->num(),this->blobs_[0]->channels(),this->blobs_[0]->height(),this->blobs_[0]->width(),bottom_diff, activation_data);
           CUDA_POST_KERNEL_CHECK; 
       

       }else{
       caffe_copy(count,bottom_diff,activation_data);
       }




   
    Threshold2<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, Dtype(0), activation_data, gates_diff);
    CUDA_POST_KERNEL_CHECK;

    caffe_gpu_asum(count,gates_diff,&diff_nonezeros);

    Threshold1<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, Dtype(0), activation_data, gates_diff);
    CUDA_POST_KERNEL_CHECK;

    caffe_gpu_asum(count,gates_diff,&diff_mean);
    
    diff_mean=diff_mean/diff_nonezeros;
    threshold=this->threshold_ratio*diff_mean;

   
    Threshold3<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, threshold, gates_diff, mutable_gates_data);
   }
 //    caffe_copy(count,top_diff,gates_diff);
//    caffe_copy(count,bottom_diff,gates_diff);
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(ReLULayer);


}  // namespace caffe
